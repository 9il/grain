#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        const char* name;
        const char* content;
        hipDrvGetErrorName(err, &name);
        hipDrvGetErrorString(err, &content);
        fprintf(stderr, "%s: %s\n", name, content);
        exit(err);
    }
}

int main(int argc, char** argv) {
    int device_id = 0;
    if (argc == 2) {
        device_id = std::stoi(argv[1]);
    }
    fprintf(stderr, "device_id: %d\n", device_id);

    hipDevice_t device;
    hipCtx_t context;
    // Initialize the driver API
    hipInit(0);
    // Get a handle to the first compute device
    hipDeviceGet(&device, device_id);
    // Create a compute device context
    hipCtxCreate(&context, 0, device);

    int devMajor, devMinor;
    checkCudaErrors(hipDeviceGetAttribute(
                        &devMajor, hipDeviceAttributeComputeCapabilityMajor, device));
    checkCudaErrors(hipDeviceGetAttribute(
                        &devMinor, hipDeviceAttributeComputeCapabilityMinor, device));
    printf("%d%d\n", devMajor, devMinor);
}